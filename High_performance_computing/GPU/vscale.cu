#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "vscale.cuh"

__global__ void vscale(const float *a, float *b, unsigned int n){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if( index < n)
  	b[index] = a[index] * b[index];
}

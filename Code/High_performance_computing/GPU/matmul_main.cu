#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "matmul.cuh"

int main(int argc, char* argv[])
{
  int n = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);

  //array initialization
  int size = n * sizeof(float);
  float* hA = (float*)malloc(size);
  float* hB = (float*)malloc(size);

 
  //initialize arrays with random float numbers [-1 1]
  const int RANGE = 1000;

  for (int i = 0; i < n; i++){
  	float randA = rand() % (RANGE + 1);
  	float randB = rand() % (RANGE + 1);
	hA[i] = randA / 500.0f-1.0f;
	hB[i] = randB / 500.0f-1.0f;
  }

  float *dA, *dB, *dC;
  hipMalloc((float**)&dA,size);
  hipMalloc((float**)&dB,size);
  hipMalloc((float**)&dC,size);
  hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
  
  const int blocksPerGrid = ( n + threads_per_block - 1 ) / threads_per_block;
    
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

    	matmul_kernel<<<blocksPerGrid, threads_per_block>>>(dA, dB, dC, n);

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Get the elapsed time in milliseconds
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	std::printf("%f\n", ms);

	hipMemcpy(hB, dB, size, hipMemcpyDeviceToHost);


  std::printf("%f\n", hB[n-1]);

  return 0;
}
